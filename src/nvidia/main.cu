#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <unistd.h>

__global__ void c_hello() { printf("Hello, World!\n"); }

int main() {
	while (true) {
		c_hello<<<1, 1>>>();
		sleep(1);
	}

	hipDeviceSynchronize();
}
